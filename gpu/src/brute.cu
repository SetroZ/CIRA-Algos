#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <map>
#include <unordered_set>
#include "/media/vetro/apps/code-projects/CIRA/code/bruteforce/cpu/src/brute.h"
using namespace std;

// CUDA constants
__constant__ double K;
__constant__ double TIME_DELAY = 1;

// GPU error check macro
#define CUDA_CHECK(call)
{
    hipError_t err = call;
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__
                  << ":" << __LINE__ << std::endl;
        exit(EXIT_FAILURE);
    }
}
__device__ double calc_time_delay_idx(double k, double DM, double d_t, double f_0, double f_1)
{
    return ((k * DM) / d_t) * (1.0 / pow(f_1, 2) - 1.0 / pow(f_0, 2));
}
__global__ void calc_paths(int min_DM, int max_DM, int d_DM, double d_t, double f_min, double f_max, double d_f,
                           int max_t_idx, int max_f_idx, PathMap *path_dict)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int DM = min_DM + idx * d_DM;
    if (DM > max_DM)
        return;

    for (double t = d_t; t <= max_t_idx * d_t; t += d_t)
    {
        double t_idx = (t - d_t) / d_t;

        for (double f_val = f_max; f_val > f_min; f_val -= d_f)
        {
            double f_low = f_val - d_f;
            double f_idx = (f_val - f_min) / d_f;

            double t_path_idx = t_idx + calc_time_delay_idx(K, DM, d_t, f_max, f_val);
            double t_path_idx_low = t_idx + calc_time_delay_idx(K, DM, d_t, f_max, f_low);

            int int_t_path_idx = round(t_path_idx);
            int int_t_path_idx_low = round(t_path_idx_low);
            int freq_value = round(f_idx);

            if (int_t_path_idx >= 0 && int_t_path_idx < max_t_idx && freq_value >= 0 && freq_value < max_f_idx)
            {
                // Store path in path_dict
                path_dict[DM][t].push_back({int_t_path_idx, freq_value});
            }
        }
    }
}
__global__ void dedisperse_kernel(const double *data, int x_size, const PathMap *path_dict, DispResults *dedispersed_results)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // TO DO
}

void dedisperse(const valarray<double> &data, const PathMap &path_dict, int x_size, DispResults &dedispersed_results)
{
    // Allocate GPU memory for data and path_dict
    double *d_data;
    CUDA_CHECK(hipMalloc(&d_data, data.size() * sizeof(double)));
    CUDA_CHECK(hipMemcpy(d_data, &data[0], data.size() * sizeof(double), hipMemcpyHostToDevice));

    PathMap *d_path_dict;
    CUDA_CHECK(hipMalloc(&d_path_dict, sizeof(PathMap)));
    CUDA_CHECK(hipMemcpy(d_path_dict, &path_dict, sizeof(PathMap), hipMemcpyHostToDevice));

    DispResults *d_dedispersed_results;
    CUDA_CHECK(hipMalloc(&d_dedispersed_results, sizeof(DispResults)));

    // Launch kernel
    int threads_per_block = 256;
    int blocks = (path_dict.size() + threads_per_block - 1) / threads_per_block;
    dedisperse_kernel<<<blocks, threads_per_block>>>(d_data, x_size, d_path_dict, d_dedispersed_results);

    // Copy results back to host
    CUDA_CHECK(hipMemcpy(&dedispersed_results, d_dedispersed_results, sizeof(DispResults), hipMemcpyDeviceToHost));

    // Free GPU memory
    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_path_dict));
    CUDA_CHECK(hipFree(d_dedispersed_results));
}