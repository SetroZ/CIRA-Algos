#include "hip/hip_runtime.h"
Path dedispersion_path(int DM, double t_0, FRBFileData *frbData, int min_t_idx, int max_t_idx, int min_f_idx, int max_f_idx)
{
    Path path;
    double t_idx = (t_0 - frbData->d_t) / frbData->d_t;
    // iterate through each frequency range from max -> min
    for (double f_val = frbData->f_max; f_val > frbData->f_min; f_val -= frbData->d_f)
    {
        double f_low = f_val - frbData->d_f;
        double f_idx = (f_val - frbData->f_min) / frbData->d_f;

        double t_path_idx = t_idx + calc_time_delay_idx(K, DM, frbData->d_t, frbData->f_max, f_val);
        double t_path_idx_low = t_idx + calc_time_delay_idx(K, DM, frbData->d_t, frbData->f_max, f_low);

        int int_t_path_idx = round(t_path_idx);
        int int_t_path_idx_low = round(t_path_idx_low);
        int freq_value = round(f_idx);
        if ((min_t_idx <= int_t_path_idx && int_t_path_idx < max_t_idx) && (min_f_idx <= freq_value && freq_value < max_f_idx))
        {
            for (int t = int_t_path_idx; t <= int_t_path_idx_low; t++)
            {
                if (min_t_idx <= t < max_t_idx)
                {
                    path.push_back({t, freq_value});
                }
            }
        }
    }
    return path;
}
