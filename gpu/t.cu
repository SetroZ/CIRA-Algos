#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include  <chrono>
// CUDA kernel for vector addition
__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

void vectorAddCPU(const std::vector<float>& A, const std::vector<float>& B, std::vector<float>& C, int N) {
    for (int i = 0; i < N; ++i) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1 << 20;  // Size of the vectors (1 million elements)
    size_t size = N * sizeof(float);

    // Host vectors
    std::vector<float> h_A(N, 1.0f);  // Initialize A with 1.0f
    std::vector<float> h_B(N, 2.0f);  // Initialize B with 2.0f
    std::vector<float> h_C(N, 0.0f);  // Vector to store the result

    // Device vectors
    float* d_A = nullptr;
    float* d_B = nullptr;
    float* d_C = nullptr;

    // Allocate memory on the device
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy vectors from host to device
    hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);

    // Timing variables
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start time
    hipEventRecord(start);

    // Launch kernel with one block per 256 threads
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Wait for the GPU to finish
    hipDeviceSynchronize();

    // Record the stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    // Copy the result back to the host
    hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);

    // Optional: Print the first few elements of the result vector
    std::cout << "First 10 elements of the result: ";
    for (int i = 0; i < 10; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // Clean up device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Clean up events
    hipEventDestroy(start);
    hipEventDestroy(stop);

      auto start_cpu = std::chrono::high_resolution_clock::now();

    // Perform vector addition on CPU
    vectorAddCPU(h_A, h_B, h_C, N);

    auto stop_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration_cpu = stop_cpu - start_cpu;

    std::cout << "CPU execution time: " << duration_cpu.count() * 1000.0f << " ms";


    return 0;
}
